// Day 2 – vectorAdd_global_variable.cu 

// nvcc -arch=sm_89 -o vectorAdd_global vectorAdd_global_variable.cu

// # default N=1M:
// ./vectorAdd_global

// # or specify size, e.g. 8M elements:
// ./vectorAdd_global 8388608

// This code performs vector addition using CUDA with global variables.
// It initializes two arrays A and B on the host, allocates memory on the device,
// and performs the addition in a kernel. The result is copied back to the host.


// Uses pinned host memory (cudaMallocHost) for higher H2D bandwidth.
// Asynchronous copies and a kernel timed via CUDA events.
// Inline error checks without macros.
// Runtime-configurable problem size (via argv[1]).


#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out of bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char* argv[]) {
    // Allow custom size via argv or default to 1M
    int N = (argc > 1) ? atoi(argv[1]) : (1 << 20);
    size_t bytes = size_t(N) * sizeof(float);

    // Pinned host allocations for higher Host to Device bandwidth
    float *h_A, *h_B, *h_C;
    hipError_t err = hipHostMalloc(&h_A, bytes, hipHostMallocDefault);
    if (err != hipSuccess) { fprintf(stderr, "Host alloc A failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }
    err = hipHostMalloc(&h_B, bytes, hipHostMallocDefault);
    if (err != hipSuccess) { fprintf(stderr, "Host alloc B failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }
    err = hipHostMalloc(&h_C, bytes, hipHostMallocDefault);
    if (err != hipSuccess) { fprintf(stderr, "Host alloc C failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }

    // Initialize
    for (int i = 0; i < N; i++) {
        h_A[i] = float(i);
        h_B[i] = float(2 * i);
    }

    // Device allocations
    float *d_A, *d_B, *d_C;
    err = hipMalloc(&d_A, bytes);
    if (err != hipSuccess) { fprintf(stderr, "hipMalloc d_A failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }
    err = hipMalloc(&d_B, bytes);
    if (err != hipSuccess) { fprintf(stderr, "hipMalloc d_B failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }
    err = hipMalloc(&d_C, bytes);
    if (err != hipSuccess) { fprintf(stderr, "hipMalloc d_C failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start
    hipEventRecord(start);

    // Async copy host to device
    err = hipMemcpyAsync(d_A, h_A, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "H2D A failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }
    err = hipMemcpyAsync(d_B, h_B, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) { fprintf(stderr, "H2D B failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }

    // Kernel launch
    const int threads = 256;
    const int blocks = (N + threads - 1) / threads;
    vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);
    err = hipGetLastError();
    if (err != hipSuccess) { fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }

    // Async copy device to host
    err = hipMemcpyAsync(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { fprintf(stderr, "D2H C failed: %s\n", hipGetErrorString(err)); return EXIT_FAILURE; }

    // Record stop & synchronize
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    // Bandwidth: 2 transfers of N floats (A and B) plus 1 transfer of N floats (C)
    double gb = double(bytes) * 3.0 / (1<<30);
    printf("VectorAdd (N=%d): Time = %.3f ms, BW = %.1f GB/s\n", N, ms, gb / (ms/1000.0));

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return EXIT_SUCCESS;
}
